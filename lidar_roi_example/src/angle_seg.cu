#include "hip/hip_runtime.h"
#include "lidar_roi_example/angle_seg.h"

#include "assert.h"
#include "ros/ros.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


namespace ROIExample
{
    inline hipError_t checkCuda(hipError_t result)
    {
        if (result != hipSuccess)
        {
            fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
            assert(result == hipSuccess);
        }
        return result;
    }

    __global__ void AzimuthSeg(pcl::PointXYZ::Ptr d_pc_in, pcl::PointXYZ::Ptr d_pc_prj)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (!(idx % 3))
        {
            float &x = (*(d_pc_in)[idx]).x;
            float &y = (*(d_pc_in)[idx]).y;
            float &z = (*(d_pc_in)[idx]).z;

            float theta = atan2(x, -y) * 180 / M_PI - 90;
            theta = (theta < 0) ? theta + 360 : theta;

            if (fabs(theta - 30) < 5)
            {
                pcl::PointXYZ point;
                point.x = x;
                point.y = y;
                point.z = z;
                d_pc_prj->push_back(point);
            }
        }
    }
    
    void AngleSeg::Init(pcl::PointCloud<pcl::PointXYZ> *&input)
    {
        auto start = ros::Time::now();

        // allocate memory for device
        int size = input->points.size();
        thrust::device_vector<pcl::PointXYZ> d_pc_in;
        thrust::device_vector<pcl::PointXYZ> d_pc_prj;

        pcl::PointCloud<pcl::PointXYZ>* pc_seg;

        // checkCuda(hipMallocManaged(&d_pc_in, sizeof(pcl::PointXYZ) * size));
        // checkCuda(hipMallocManaged(&d_pc_prj, sizeof(pcl::PointXYZ) * size));

        // copy pointcloud data to device
        checkCuda(hipMemcpy(d_pc_in, input->points, sizeof(pcl::PointXYZ) * size, hipMemcpyHostToDevice));
        pcl::PointXYZ::Ptr d_pc_ptr = thrust::raw_pointer_cast(d_pc_in.data);
        pcl::PointXYZ::Ptr d_pc_prj_ptr = thrust::raw_pointer_cast(d_pc_prj.data);

        // 170 blocks, 1020 threads
        // 1020/3 = 340 points per block
        // 170 x 340 = 57800 maximum points
        AzimuthSeg<<<170, 1020>>>(d_pc_in, d_pc_prj);

        checkCuda(hipMemcpy(pc_seg->points, d_pc_prj, sizeof(pcl::PointXYZ) * size, hipMemcpyHostToDevice));
        pc_seg_ = pc_seg;

        auto end = ros::Time::now();
        ROS_WARN("time %f", (end - start).toSec());
    }

    pcl::PointCloud<pcl::PointXYZ>* AngleSeg::GetPC()
    {
        return pc_seg_;
    }
}